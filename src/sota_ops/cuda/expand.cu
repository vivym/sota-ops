#include <limits>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/scatter.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

#include "sota_ops/expand.h"
#include "sota_ops/utils/thrust_allocator.h"

namespace sota_ops::expand {

template <typename index_t>
void expand_csr_cuda_impl(
    at::Tensor& output,
    at::Tensor& sizes,
    const at::Tensor& offsets,
    int64_t output_size) {
  auto stream = at::cuda::getCurrentCUDAStream().stream();
  auto policy = thrust::cuda::par(utils::ThrustAllocator()).on(stream);

  index_t num_segments = offsets.size(0) - 1;

  auto output_ptr = output.data_ptr<index_t>();
  auto offsets_ptr = offsets.data_ptr<index_t>();
  auto sizes_ptr = sizes.data_ptr<index_t>();

  thrust::transform(
      policy,
      thrust::make_zip_iterator(thrust::make_tuple(
          offsets_ptr, offsets_ptr + 1)),
      thrust::make_zip_iterator(thrust::make_tuple(
          offsets_ptr + num_segments, offsets_ptr + 1 + num_segments)),
      sizes_ptr,
      [=] __host__ __device__ (thrust::tuple<index_t, index_t> t) {
        return thrust::get<1>(t) - thrust::get<0>(t);
      });

  thrust::scatter_if(
      policy,
      thrust::counting_iterator<index_t>(0),
      thrust::counting_iterator<index_t>(num_segments),
      offsets_ptr,
      sizes_ptr,
      output_ptr);

  thrust::inclusive_scan(
      policy,
      output_ptr,
      output_ptr + output_size,
      output_ptr,
      thrust::maximum<index_t>());
}

std::tuple<at::Tensor, at::Tensor> expand_csr_cuda(const at::Tensor& offsets, int64_t output_size) {
  TORCH_CHECK(offsets.is_cuda(), "offsets must be a CUDA tensor");
  TORCH_CHECK(offsets.dim() == 1, "offsets must be a 1D tensor");
  TORCH_CHECK(offsets.is_contiguous(), "offsets must be contiguous");

  auto num_segments = offsets.size(0) - 1;
  auto output = at::zeros({output_size}, offsets.options());
  auto sizes = at::empty({num_segments}, offsets.options());

  if (offsets.scalar_type() == at::kInt) {
    expand_csr_cuda_impl<int32_t>(output, sizes, offsets, output_size);
  } else if (offsets.scalar_type() == at::kLong) {
    expand_csr_cuda_impl<int64_t>(output, sizes, offsets, output_size);
  } else {
    AT_ERROR("Unsupported type (expand_csr_cuda)");
  }

  return {output, sizes};
}

TORCH_LIBRARY_IMPL(sota_ops, CUDA, m) {
  m.impl(TORCH_SELECTIVE_NAME("sota_ops::expand_csr"),
         TORCH_FN(expand_csr_cuda));
}

} // namespace sota_ops::expand
